#include "hip/hip_runtime.h"
#include "spmv.h"

// Advanced SpMV kernels for computing  y = A * x, where A is a sparse matrix
// represented by different formats.


/**
 * Sell-C-sigma implementation templated by C.
 */
template <ST C, typename VT, typename IT>
__device__
static void
scs_impl(const ST n_chunks,
         const IT * RESTRICT chunk_ptrs,
         const IT * RESTRICT chunk_lengths,
         const IT * RESTRICT col_idxs,
         const VT * RESTRICT values,
         const VT * RESTRICT x,
         VT * RESTRICT y)
{
    ST row = threadIdx.x + blockDim.x * blockIdx.x;
    ST c   = row / C;  // the no. of the chunk
    ST idx = row % C;  // index inside the chunk

    if (row < n_chunks * C) {
        VT tmp{};
        IT cs = chunk_ptrs[c];

        for (ST j = 0; j < chunk_lengths[c]; ++j) {
            tmp += values[cs + j * C + idx] * x[col_idxs[cs + j * C +idx]];
        }

        y[row] += tmp;
    }

}


/**
 * Dispatch to Sell-C-sigma kernels templated by C.
 *
 * Note: only works for selected Cs, see INSTANTIATE_CS.
 */
template <typename VT, typename IT>
__global__
static void
spmv_scs_c(
             const ST C,
             const ST n_chunks,
             const IT * RESTRICT chunk_ptrs,
             const IT * RESTRICT chunk_lengths,
             const IT * RESTRICT col_idxs,
             const VT * RESTRICT values,
             const VT * RESTRICT x,
             VT * RESTRICT y)
{
    switch (C)
    {
        #define INSTANTIATE_CS X(2) X(4) X(8) X(16) X(32) X(64) X(128)

        #define X(CC) case CC: scs_impl<CC>(n_chunks, chunk_ptrs, chunk_lengths, col_idxs, values, x, y); break;
        INSTANTIATE_CS
        #undef X

#ifdef SCS_C
    case SCS_C:
        case SCS_C: scs_impl<SCS_C>(n_chunks, chunk_ptrs, chunk_lengths, col_idxs, values, x, y);
        break;
#endif
    default:
        //fprintf(stderr,
        //        "ERROR: for C=%ld no instantiation of a sell-c-sigma kernel exists.\n",
        //        long(C));
        // exit(EXIT_FAILURE);
    }
}

//          name     function        is_gpu  format
REG_KERNELS("scs-c", spmv_scs_c,     true,   MatrixFormat::SellCSigma);

